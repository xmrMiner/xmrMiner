#include <ctype.h>
#include <unistd.h>
#include <stdio.h>
#include <stdint.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

extern "C"
{
#include "xmrMiner-config.h"
#include "miner.h"
}
#include "cryptonight.h"
#include "cuda_device.hpp"

extern "C"
{
    extern char *device_name[8];
    extern int device_arch[8][2];
    extern int device_mpcount[8];
    extern int device_map[8];
    extern int device_config[8][2];
}

// Zahl der CUDA Devices im System bestimmen
extern "C" int cuda_num_devices()
{
	int version;
	hipError_t err = hipDriverGetVersion(&version);
	if(err != hipSuccess)
	{
		applog(LOG_ERR, "Unable to query CUDA driver version! Is an nVidia driver installed?");
		exit(1);
	}

	if(version < CUDART_VERSION)
	{
		applog(LOG_ERR, "Driver does not support CUDA %d.%d API! Update your nVidia driver!", CUDART_VERSION / 1000, (CUDART_VERSION % 1000) / 10);
		exit(1);
	}

	int GPU_N;
	err = hipGetDeviceCount(&GPU_N);
	if(err != hipSuccess)
	{
		if(err != hipErrorNoDevice)
			applog(LOG_ERR, "No CUDA device found!");
		else
			applog(LOG_ERR, "Unable to query number of CUDA devices!");
		exit(1);
	}
	return GPU_N;
}

extern "C" void cuda_deviceinfo()
{
	hipError_t err;
	int GPU_N;
	err = hipGetDeviceCount(&GPU_N);
	if(err != hipSuccess)
	{
		if(err != hipErrorNoDevice)
			applog(LOG_ERR, "No CUDA device found!");
		else
			applog(LOG_ERR, "Unable to query number of CUDA devices!");
		exit(1);
	}

	for(int i = 0; i < GPU_N; i++)
	{
		hipDeviceProp_t props;
		hipError_t err = hipGetDeviceProperties(&props, device_map[i]);
		if(err != hipSuccess)
		{
			printf("\nGPU %d: %s\n%s line %d\n", device_map[i], hipGetErrorString(err), __FILE__, __LINE__);
			exit(1);
		}

		device_name[i] = strdup(props.name);
		device_mpcount[i] = props.multiProcessorCount;
		device_arch[i][0] = props.major;
		device_arch[i][1] = props.minor;
	}
}

static bool substringsearch(const char *haystack, const char *needle, int &match)
{
	int hlen = (int)strlen(haystack);
	int nlen = (int)strlen(needle);
	for(int i = 0; i < hlen; ++i)
	{
		if(haystack[i] == ' ') continue;
		int j = 0, x = 0;
		while(j < nlen)
		{
			if(haystack[i + x] == ' ')
			{
				++x; continue;
			}
			if(needle[j] == ' ')
			{
				++j; continue;
			}
			if(needle[j] == '#') return ++match == needle[j + 1] - '0';
			if(tolower(haystack[i + x]) != tolower(needle[j])) break;
			++j; ++x;
		}
		if(j == nlen) return true;
	}
	return false;
}

extern "C" int cuda_finddevice(char *name)
{
	int num = cuda_num_devices();
	int match = 0;
	for(int i = 0; i < num; ++i)
	{
		hipDeviceProp_t props;
		if(hipGetDeviceProperties(&props, i) == hipSuccess)
			if(substringsearch(props.name, name, match)) return i;
	}
	return -1;
}

static uint32_t *d_long_state[8];
static uint32_t *d_ctx_state[8];
static uint32_t *d_ctx_a[8];
static uint32_t *d_ctx_b[8];
static uint32_t *d_ctx_key1[8];
static uint32_t *d_ctx_key2[8];
static uint32_t *d_ctx_text[8];

extern "C"
{
extern bool opt_benchmark;
}

extern "C" void cryptonight_hash(void* output, const void* input, size_t len);

extern "C" int scanhash_cryptonight(int thr_id, uint32_t *pdata, int dlen, const uint32_t *ptarget, uint32_t max_nonce, unsigned long *hashes_done, uint32_t *results)
{
	hipError_t err;
	int res;
	uint32_t *nonceptr = (uint32_t*)(((char*)pdata) + 39);
	const uint32_t first_nonce = *nonceptr;
	uint32_t nonce = *nonceptr;
	int cn_blocks = device_config[thr_id][0];
	int cn_threads = device_config[thr_id][1];
	if(opt_benchmark)
	{
		((uint32_t*)ptarget)[7] = 0x0000ff;
		pdata[17] = 0;
	}
	const uint32_t Htarg = ptarget[7];
	const uint32_t throughput = cn_threads * cn_blocks;
	if(sizeof(size_t) == 4 && throughput > 0xffffffff / MEMORY)
	{
		applog(LOG_ERR, "GPU %d: THE 32bit VERSION CAN'T ALLOCATE MORE THAN 4GB OF MEMORY!", device_map[thr_id]);
		applog(LOG_ERR, "GPU %d: PLEASE REDUCE THE NUMBER OF THREADS OR BLOCKS", device_map[thr_id]);
		exit(1);
	}
	const size_t alloc = (size_t)MEMORY * throughput;

	static bool init[8] = {false, false, false, false, false, false, false, false};
	if(!init[thr_id])
	{
		err = hipSetDevice(device_map[thr_id]);
		if(err != hipSuccess)
		{
			applog(LOG_ERR, "GPU %d: %s", device_map[thr_id], hipGetErrorString(err));
		}
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		hipMalloc(&d_long_state[thr_id], alloc);
		exit_if_cudaerror(thr_id, __FILE__, __LINE__);
		hipMalloc(&d_ctx_state[thr_id], 50 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FILE__, __LINE__);
		hipMalloc(&d_ctx_key1[thr_id], 40 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FILE__, __LINE__);
		hipMalloc(&d_ctx_key2[thr_id], 40 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FILE__, __LINE__);
		hipMalloc(&d_ctx_text[thr_id], 32 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FILE__, __LINE__);
		hipMalloc(&d_ctx_a[thr_id], 4 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FILE__, __LINE__);
		hipMalloc(&d_ctx_b[thr_id], 4 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FILE__, __LINE__);

		cryptonight_extra_cpu_init(thr_id);

		init[thr_id] = true;
	}

	cryptonight_extra_cpu_setData(thr_id, (const void *)pdata, dlen, (const void *)ptarget);

	do
	{
		uint32_t foundNonce[2];

		cryptonight_extra_cpu_prepare(thr_id, throughput, dlen, nonce, d_ctx_state[thr_id], d_ctx_a[thr_id], d_ctx_b[thr_id], d_ctx_key1[thr_id], d_ctx_key2[thr_id]);
		cryptonight_core_cpu_hash(thr_id, cn_blocks, cn_threads, d_long_state[thr_id], d_ctx_state[thr_id], d_ctx_a[thr_id], d_ctx_b[thr_id], d_ctx_key1[thr_id], d_ctx_key2[thr_id]);
		cryptonight_extra_cpu_final(thr_id, throughput, nonce, foundNonce, d_ctx_state[thr_id]);

		if(foundNonce[0] < 0xffffffff)
		{
			uint32_t vhash64[8] = {0, 0, 0, 0, 0, 0, 0, 0};
			uint32_t tempdata[32];
			uint32_t *tempnonceptr = (uint32_t*)(((char*)tempdata) + 39);
			*tempnonceptr = foundNonce[0];
			memcpy(tempdata, pdata, dlen);
			cryptonight_hash(vhash64, tempdata, dlen);
			if((vhash64[7] <= Htarg) && fulltest(vhash64, ptarget))
			{
				res = 1;
				results[0] = foundNonce[0];
				*hashes_done = nonce - first_nonce + throughput;
				if(foundNonce[1] < 0xffffffff)
				{
					*tempnonceptr = foundNonce[1];
					cryptonight_hash(vhash64, tempdata, dlen);
					if((vhash64[7] <= Htarg) && fulltest(vhash64, ptarget))
					{
						res++;
						results[1] = foundNonce[1];
					}
					else
					{
						applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", device_map[thr_id], foundNonce[1]);
					}
				}
				return res;
			}
			else
			{
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", device_map[thr_id], foundNonce[0]);
			}
		}
		if((nonce & 0x00ffffff) > (0x00ffffff - throughput))
			nonce = max_nonce;
		else
			nonce += throughput;
	} while(nonce < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = nonce - first_nonce;
	return 0;
}
